#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

float gpu_mm(double* A_dev, double* B_dev, double* C_dev,double* C_star,int* n_loc_vect,int n_loc,int i,int* n_col_sum,double alpha,double beta, int N, int n_prc, hipblasHandle_t handle) {

     hipEvent_t start, stop;
     float time;
     hipEventCreate(&start);
     hipEventCreate(&stop);
     
     hipEventRecord(start, 0);
     hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n_loc_vect[i], n_loc, N, &alpha, B_dev, n_loc_vect[i],A_dev, N, &beta, &C_dev[n_col_sum[i]], N);
     hipEventRecord(stop, 0);

     hipEventSynchronize(stop);
     hipEventElapsedTime(&time, start, stop);
 
     return time;
}